#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <vector>
#include <iostream>
#include <ctime>
#include <chrono>

#define threadsperblock 1024

// Kernel to initialize CURAND states
__global__ void InitCurandStates(hiprandState* states, long long seed, int NumberOfElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NumberOfElements) {
        hiprand_init(seed, tid, 0, &states[tid]);
    }
}

// Kernel to generate random numbers
__global__ void GenerateRandomArrayKernel(int* d_array, hiprandState* states, int lowerBound, int upperBound, int NumberOfElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NumberOfElements) {
        hiprandState localState = states[tid];
        float randomValue = hiprand_uniform(&localState); // Generate random float
        d_array[tid] = lowerBound + (int)((upperBound - lowerBound + 1) * randomValue);
        states[tid] = localState;
    }
}

int* CreateRandomArray(int NumberOfElements, int lowerBound, int upperBound) {
    int* d_array;
    int* h_array = new int[NumberOfElements];
    hiprandState* d_states;

    hipMalloc(&d_array, sizeof(int) * NumberOfElements);
    hipMalloc(&d_states, sizeof(hiprandState) * NumberOfElements);

    int blocksPerGrid = (NumberOfElements + threadsperblock - 1) / threadsperblock;
    long long seed = time(0);

    InitCurandStates << <blocksPerGrid, threadsperblock >> > (d_states, seed, NumberOfElements);
    GenerateRandomArrayKernel << <blocksPerGrid, threadsperblock >> > (d_array, d_states, lowerBound, upperBound, NumberOfElements);

    hipMemcpy(h_array, d_array, sizeof(int) * NumberOfElements, hipMemcpyDeviceToHost);

    hipFree(d_array);
    hipFree(d_states);

    return h_array;
}

// CUDA Counting Sort Kernels
__global__ void countKernel(const int* inputVector, int* countArray, long int NumberOfElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < NumberOfElements) {
        atomicAdd(&countArray[inputVector[idx]], 1);
    }
}

__global__ void placeKernel(const int* inputVector, int* countArray, int* outputArray, long int NumberOfElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < NumberOfElements) {
        int value = inputVector[idx];
        int pos = atomicSub(&countArray[value], 1) - 1;
        outputArray[pos] = value;
    }
}

void countingSortCUDA(int upperBound, int NumberOfElements, std::vector<int>& inputVector) {
    int* d_inputVector, * d_countArray, * d_outputArray;
    const int range = upperBound + 1;

    hipMalloc(&d_inputVector, NumberOfElements * sizeof(int));
    hipMalloc(&d_outputArray, NumberOfElements * sizeof(int));
    hipMalloc(&d_countArray, range * sizeof(int));



    hipMemset(d_countArray, 0, range * sizeof(int));
    hipMemcpy(d_inputVector, inputVector.data(), NumberOfElements * sizeof(int), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    int blocksPerGrid = (NumberOfElements + threadsperblock - 1) / threadsperblock;

    countKernel << <blocksPerGrid, threadsperblock >> > (d_inputVector, d_countArray, NumberOfElements);

    thrust::device_ptr<int> thrust_countArray(d_countArray);
    thrust::inclusive_scan(thrust_countArray, thrust_countArray + range, thrust_countArray);

    placeKernel << <blocksPerGrid, threadsperblock >> > (d_inputVector, d_countArray, d_outputArray, NumberOfElements);


    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start;
    std::cout << "Time taken to sort the list: " << duration.count() << " milliseconds" << std::endl;

    hipMemcpy(inputVector.data(), d_outputArray, NumberOfElements * sizeof(int), hipMemcpyDeviceToHost);


    hipFree(d_inputVector);
    hipFree(d_countArray);
    hipFree(d_outputArray);
}

int main() {
    const int NumberOfElements = 1L << 28;  // Maximum 21
    const int lowerBound = 1;
    const int upperBound = 1 << 12;

    int* h_randomList = CreateRandomArray(NumberOfElements, lowerBound, upperBound);
    std::vector<int> inputVector(h_randomList, h_randomList + NumberOfElements);

    std::cout << "Sorting " << NumberOfElements << " elements..." << std::endl;

    countingSortCUDA(upperBound, NumberOfElements, inputVector);

    // Verify the array is sorted
    for (int i = 1; i < NumberOfElements - 1; i++) {
        if (!(inputVector[i - 1] <= inputVector[i])) {
            std::cerr << "Array is not sorted correctly!" << std::endl;
            std::cout << "Error at index " << i << ": " << inputVector[i - 1] << " > " << inputVector[i] << std::endl;
            delete[] h_randomList;
            return -1;
        }
    }
    std::cout << "Array is sorted correctly!" << std::endl;

    delete[] h_randomList;
    return 0;
}
